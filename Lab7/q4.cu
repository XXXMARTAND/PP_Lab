#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void sin(float *a, float *b)
{
	int i = blockIdx.x;
	b[i] = sinf(a[i]);
}

int main()
{
	float a[20], b[20];
	int n, i;
	printf("Enter size");
	scanf("%d", &n);
	printf("\nEnter rad array \n");

	for(i = 0; i < n; i++)
		scanf("%f", &a[i]);

	float *d_a, *d_b;
	int size = sizeof(float) * 20;
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);

	sin<<<n,1>>>(d_a, d_b);
	hipMemcpy(&b, d_b, size, hipMemcpyDeviceToHost);
	
	printf("\nResultant arr \n");
		for(i = 0; i < n; i++)
			printf("%f   ", b[i]);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}
