#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void cal(int *a, int *b, int x1)
{
	int i = blockIdx.x;
	b[i] = x1 * a[i] + b[i];
}

int main()
{
	int a[20], b[20];
	int n, i;

	printf("Enter size");
	scanf("%d",&n);
	
	printf("\nEnter set 1 \n");
	for(i = 0; i < n; i++)
		scanf("%d", &a[i]);
	printf("Enter set 2 \n");
	for(i = 0; i < n; i++)
		scanf("%d", &b[i]);

	int alpha;
	printf("Enter AA ");
	scanf("%d", &alpha);

	int *d_a, *d_b, *d_c, *d_d, *d_e;
	int size = sizeof(int) * 20;
	
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	hipMalloc((void**)&d_d, size);
	hipMalloc((void**)&d_e, size);
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
	cal<<<n, 1>>>(d_a, d_b, alpha);
	
	hipMemcpy(&b, d_b, size, hipMemcpyDeviceToHost);
	printf("\nExpansion res \n");
		for(i = 0; i < n; i++)
			printf("%d  ", b[i]);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}