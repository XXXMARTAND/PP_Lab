#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

//function definition 

__global__ void Fun1(int *a, int *b, int *c)
{
	int i = blockIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void Fun2(int *a, int *b, int *c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void Fun3(int *a, int *b, int *c, int n)
{
	int idx = threadIdx.x;
	int id = blockIdx.x * blockDim.x;
	idx += id;

	if(idx < n)
		c[idx] = a[idx] + b[idx];
}


//program
int main()
{
	int a[20], b[20], c[20], n, i;
	printf("Enter n ");
	scanf("%d", &n);
	printf("\nEnter set 1\n");
	for(i = 0; i < n; i++)
		scanf("%d", &a[i]);
	printf("Enter set 2\n");
	for(i = 0; i < n; i++)
		scanf("%d", &b[i]);

	int *d_a, *d_b, *d_c, *d_d, *d_e;

	int size = sizeof(int) * 20;
	
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	hipMalloc((void**)&d_d, size);
	hipMalloc((void**)&d_e, size);

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);

	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	//func def
	Fun1<<<n, 1>>>(d_a, d_b, d_c);
	
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("Res 1\n");
	for(i = 0; i < n; i++)
		printf("%d  \n", c[i]);

	int d[20];
	//func def
	Fun2<<<1, n>>>(d_a, d_b, d_d);
	
	hipMemcpy(&d, d_d, size, hipMemcpyDeviceToHost);
	printf("Res 2\n");
	for(i = 0; i < n; i++)
		printf("%d  \n", d[i]);
	//init
	int e[20];

	int thread = 256;

	int xyz = (int)(n / thread);
	//func def
	Fun3<<<xyz, 256>>>(d_a, d_b, d_e, n);
	
	hipMemcpy(&e, d_e, size, hipMemcpyDeviceToHost);
	printf("Res 3\n");

	for(i = 0; i < n; i++)
		printf("%d \n", e[i]);
	
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_d);
	
	return 0;
}